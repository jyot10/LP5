#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add(int *a, int *b, int *c, int n){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index < n)
        c[index] = a[index] + b[index];
}

int main() {
    int n = 512;
    int size = n * sizeof(int);
    int *h_a = (int*)malloc(size);
    int *h_b = (int*)malloc(size);
    int *h_c = (int*)malloc(size);

    for(int i = 0; i < n; i++){
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    add<<<(n+255)/256, 256>>>(d_a, d_b, d_c, n);

    // Error check
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("CUDA Error: %s\n", hipGetErrorString(err));

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    for(int i = 0; i < 10; i++){
        printf("%d + %d = %d\n", h_a[i], h_b[i], h_c[i]);
    }

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    free(h_a); free(h_b); free(h_c);

    return 0;
}
